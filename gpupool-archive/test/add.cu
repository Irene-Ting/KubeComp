
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(int argc, char** argv) 
{
  if (argc != 2) 
  {
    fprintf(stderr, "must provide exactly 1 argument!\n");
    return 1;
  }

  unsigned long long sec = atoll(argv[1]);

  std::cout << "init" << std::endl;
  
  int N = 1<<10;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on the GPU
  std::cout << "Start loop for " << sec << "s" << std::endl;
  auto start = std::chrono::steady_clock::now();
  while (std::chrono::steady_clock::now() - start < std::chrono::seconds(sec)) {
    add<<<1, 1>>>(N, x, y);
    hipDeviceSynchronize();
  } 
  std::cout << "End loop" << std::endl;

  // Wait for GPU to finish before accessing on host
  // cudaDeviceSynchronize();

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}