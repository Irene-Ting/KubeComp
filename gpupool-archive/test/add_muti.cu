
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>
#include <omp.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(int argc, char** argv) 
{
    if (argc != 3) 
    {
        fprintf(stderr, "must provide exactly 1 argument!\n");
        return 1;
    }

    unsigned long long num_of_gpu = atoll(argv[1]);
    unsigned long long sec = atoll(argv[2]);

    std::cout << "init " << num_of_gpu << " gpus" << std::endl;
    
    int N = 1<<10;
    float *x = (float*)malloc(N*sizeof(float));
    float *y = (float*)malloc(N*sizeof(float));
    float *x_device[8];
    float *y_device[8];

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
  
    #pragma omp parallel num_threads(num_of_gpu)
    {
        int dev_id = omp_get_thread_num();
        hipSetDevice(dev_id);

        hipMalloc(&x_device[dev_id], N * sizeof(float));
        hipMemcpy(x_device[dev_id], x, N * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc(&y_device[dev_id], N * sizeof(float));
        hipMemcpy(y_device[dev_id], y, N * sizeof(float), hipMemcpyHostToDevice);

        // Run kernel on the GPU
        if (dev_id == 0)
            std::cout << "Start loop for " << sec << "s" << std::endl;
        auto start = std::chrono::steady_clock::now();
        while (std::chrono::steady_clock::now() - start < std::chrono::seconds(sec)) {
            add<<<1, 1>>>(N, x_device[dev_id], y_device[dev_id]);
            hipDeviceSynchronize();
        } 

        #pragma omp barrier
        if (dev_id == 0)
            std::cout << "End loop" << std::endl;

        // Free memory
        hipFree(x_device[dev_id]);
        hipFree(y_device[dev_id]);
    }
    
    return 0; 
}

// nvcc add_muti.cu -Xcompiler -fopenmp -o add_multi